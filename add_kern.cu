
#include <hip/hip_runtime.h>
__global__ 
void add(double* out,const double* a,const double* b){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    out[idx]=a[idx]+b[idx];
}